#include "hip/hip_runtime.h"

#include "myutils.h"


//矩阵生成
void generateNormalMatrix(double *dA, long int m, long int n, double mean, double stddev)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateNormalDouble(gen, dA, long(m * n), mean, stddev);
}

//矩阵上三角拷贝
__global__ void getU(int m, int n, double *A, int ldA, double *U, int ldU)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        if (i > j)
            U[i + j * ldU] = 0;
        else
            U[i + j * ldU] = A[i + j * ldA];
    }
}

void launchKernel_getU(dim3 gridDim, dim3 blockDim, int m, int n, double *A, int ldA, double *U, int ldU)
{
    getU<<<gridDim, blockDim>>>(m, n, A, ldA, U, ldU);
}

//正交性和后向误差检测
void checkBackwardError(int m, int n, double *A, int lda, double *Q, int ldq, double *R, int ldr)  
{  
    double normA = Dnorm(m, n, A);   
    // printf("normA: %f\n", normA);
    double alpha = 1.0;  
    double beta = -1.0;  

    dgemm(m, n, n, Q, ldq, R, ldr, A, lda, alpha, beta);  

    // 计算 ||A - QR|| / ||A||  
    double normRes = Dnorm(m, n, A);  
    // printf("normRes: %f\n", normRes);
    printf("Backward error: ||A-QR||/(||A||) = %.6e\n", normRes / normA);  
}

//二范数
double Dnorm(long int m, long int n, double *dA) {  
    hipblasHandle_t handle;  
    hipblasCreate(&handle);  
    double dn;  
    int incx = 1;  
    hipblasDnrm2(handle, m * n, dA, incx, &dn);  
    hipblasDestroy(handle);  
    return dn;  
}
// 矩阵乘法  
//C = AB + C
void dgemm(int m, int n, int k, double *dA, int lda, double *dB, int ldb, double *dC, int ldc, double alpha, double beta)  
{  
    hipblasHandle_t handle;  
    hipblasCreate(&handle);  
    double done = alpha;  
    double dzero = beta;  
    hipblasStatus_t status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,  
                                        m, n, k,  
                                        &done, dA, lda,  
                                        dB, ldb,  
                                        &dzero, dC, ldc);  
    if (status != HIPBLAS_STATUS_SUCCESS) {  
        fprintf(stderr, "CUBLAS Dgemm failed\n");  
    }  
    hipblasDestroy(handle);  
}

void checkOtho(long int m, long int n, double *Q, int ldq) {  
    double *I;  
    hipMalloc(&I, sizeof(double) * n * n);  

    // Define grid and block sizes  
    dim3 grid((n + 15) / 16, (n + 15) / 16);  
    dim3 block(16, 16);  

    // Generate the identity matrix on the device  
    setEye<<<grid, block>>>(I, n);  
    hipDeviceSynchronize();  

    double dnegone = -1.0;  
    double done = 1.0;  

    hipblasHandle_t handle;  
    hipblasCreate(&handle);  

    // Compute I - Q^T * Q  
    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m,  
                &dnegone, Q, HIP_R_64F, ldq, Q, HIP_R_64F, ldq,  
                &done, I, HIP_R_64F, n, HIP_R_64F,  
                HIPBLAS_GEMM_DEFAULT);  
    // Compute the norm of (I - Q^T * Q)  
    double normRes = Dnorm(n, n, I);  
    printf("orthogonal error: ||I - Q'*Q||/N = %.6e\n", normRes / n);  
    hipFree(I);  
    hipblasDestroy(handle);  
}

__global__ void setEye(double *I, long int n) {  
    // 获取当前线程的行和列索引  
    long int row = blockIdx.y * blockDim.y + threadIdx.y;  // 行索引  
    long int col = blockIdx.x * blockDim.x + threadIdx.x;  // 列索引  

    // 确保线程索引在矩阵维度范围内  
    if (row < n && col < n) {  
        if (row == col) {  
            I[row * n + col] = 1.0;  // 对角线元素设为1  
        } else {  
            I[row * n + col] = 0.0;  // 其他元素设为0  
        }  
    }  
}  

__global__ void setEye(double *I, long int m, long int n) {  
    // 获取当前线程的行和列索引  
    long int row = blockIdx.y * blockDim.y + threadIdx.y;  // 行索引  
    long int col = blockIdx.x * blockDim.x + threadIdx.x;  // 列索引  

    // 确保线程索引在矩阵维度范围内  
    if (row < m && col < n) {  
        if (row == col) {  
            I[row * n + col] = 1.0;  // 对角线元素设为1  
        } else {  
            I[row * n + col] = 0.0;  // 其他元素设为0  
        }  
    }  
}

//计算I - Q
__global__ void IminusQ(long m, long n, double *Q, long ldq)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("come %d, %d, %d,\n", __LINE__, i, j);
    // __syncthreads();

    if (i < m && j < n)
    {
        if (i == j)
        {
            Q[i + j * ldq] = 1.0 - Q[i + j * ldq];
        }
        else
        {
            Q[i + j * ldq] = -Q[i + j * ldq];
        }

        // printf("come %d, %d, %d,\n", __LINE__, i, j);
        // __syncthreads();
    }
}


//float版本的实现

//矩阵生成  
void generateNormalMatrix(float *dA, long int m, long int n, float mean, float stddev)  
{  
    hiprandGenerator_t gen;  
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);  
    int seed = 3000;  
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);  

    hiprandGenerateNormal(gen, dA, long(m * n), mean, stddev);  
}  

//矩阵上三角拷贝  
__global__ void getU(int m, int n, float *A, int ldA, float *U, int ldU)  
{  
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;  
    if (i < m && j < n)  
    {  
        if (i > j)  
            U[i + j * ldU] = 0.0f;  
        else  
            U[i + j * ldU] = A[i + j * ldA];  
    }  
}  

void launchKernel_getU(dim3 gridDim, dim3 blockDim, int m, int n, float *A, int ldA, float *U, int ldU)  
{  
    getU<<<gridDim, blockDim>>>(m, n, A, ldA, U, ldU);  
}  

//正交性和后向误差检测  
void checkBackwardError(int m, int n, float *A, int lda, float *Q, int ldq, float *R, int ldr)  
{  
    float normA = Snorm(m, n, A);   
    float alpha = 1.0f;  
    float beta = -1.0f;  

    sgemm(m, n, n, Q, ldq, R, ldr, A, lda, alpha, beta);  

    float normRes = Snorm(m, n, A);  
    printf("Backward error: ||A-QR||/(||A||) = %.6e\n", normRes / normA);  
}  

//二范数  
float Snorm(long int m, long int n, float *dA) {  
    hipblasHandle_t handle;  
    hipblasCreate(&handle);  
    float sn;  
    int incx = 1;  
    hipblasSnrm2(handle, m * n, dA, incx, &sn);  
    hipblasDestroy(handle);  
    return sn;  
}  

// 矩阵乘法  
void sgemm(int m, int n, int k, float *dA, int lda, float *dB, int ldb, float *dC, int ldc, float alpha, float beta)  
{  
    hipblasHandle_t handle;  
    hipblasCreate(&handle);  
    float done = alpha;  
    float dzero = beta;  
    hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,  
                                        m, n, k,  
                                        &done, dA, lda,  
                                        dB, ldb,  
                                        &dzero, dC, ldc);  
    if (status != HIPBLAS_STATUS_SUCCESS) {  
        fprintf(stderr, "CUBLAS Sgemm failed\n");  
    }  
    hipblasDestroy(handle);  
}  

void checkOtho(long int m, long int n, float *Q, int ldq) {  
    float *I;  
    hipMalloc(&I, sizeof(float) * n * n);  

    dim3 grid((n + 15) / 16, (n + 15) / 16);  
    dim3 block(16, 16);  

    setEye<<<grid, block>>>(I, n);  
    hipDeviceSynchronize();  

    float snegone = -1.0f;  
    float sone = 1.0f;  

    hipblasHandle_t handle;  
    hipblasCreate(&handle);  

    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m,  
                &snegone, Q, HIP_R_32F, ldq, Q, HIP_R_32F, ldq,  
                &sone, I, HIP_R_32F, n, HIP_R_32F,  
                HIPBLAS_GEMM_DEFAULT);  

    float normRes = Snorm(n, n, I);  
    printf("orthogonal error: ||I - Q'*Q||/N = %.6e\n", normRes / n);  
    hipFree(I);  
    hipblasDestroy(handle);  
}  

__global__ void setEye(float *I, long int n) {  
    long int row = blockIdx.y * blockDim.y + threadIdx.y;  
    long int col = blockIdx.x * blockDim.x + threadIdx.x;  

    if (row < n && col < n) {  
        if (row == col) {  
            I[row * n + col] = 1.0f;  
        } else {  
            I[row * n + col] = 0.0f;  
        }  
    }  
}  

__global__ void setEye(float *I, long int m, long int n) {  
    long int row = blockIdx.y * blockDim.y + threadIdx.y;  
    long int col = blockIdx.x * blockDim.x + threadIdx.x;  

    if (row < m && col < n) {  
        if (row == col) {  
            I[row * n + col] = 1.0f;  
        } else {  
            I[row * n + col] = 0.0f;  
        }  
    }  
}  

//计算I - Q  
__global__ void IminusQ(long m, long n, float *Q, long ldq)  
{  
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    int j = blockIdx.y * blockDim.y + threadIdx.y;  

    if (i < m && j < n)  
    {  
        if (i == j)  
        {  
            Q[i + j * ldq] = 1.0f - Q[i + j * ldq];  
        }  
        else  
        {  
            Q[i + j * ldq] = -Q[i + j * ldq];  
        }  
    }  
}  
// 将矩阵输出到文件中，支持文件追加写入  
void writeMatrixToCsvV2(float *dA, long ldA, long rows, long cols, const std::string &fileName, const std::string &matrixName)  
{  
    float matrix;  

    // 以 "追加写入" 模式打开文件  
    std::ofstream file(fileName, std::ios::app);  

    if (file.is_open())  
    {  
        // 如果需要，每次追加写入前可标明当前矩阵的名称  
        file << matrixName << std::endl;  

        // 设置浮点数输出固定为 4 位小数 + 列对齐  
        file << std::fixed << std::setprecision(3);  

        // 假设每列宽度固定为 10，可以调整此值以适配更宽或更小的输出  
        const int columnWidth = 6;  

        for (long i = 0; i < rows; i++)  
        {  
            for (long j = 0; j < cols; j++)  
            {  
                // 从设备内存中复制一个元素到主机  
                hipMemcpy(&matrix, dA + i + j * ldA, sizeof(float), hipMemcpyDeviceToHost);  

                // 输出当前元素，使用宽度和右对齐  
                file << std::right << std::setw(columnWidth) << matrix;  

                // 如果不是当前行的最后一个元素，添加逗号对齐  
                if ((cols - 1) != j)  
                {  
                    file << " ";  
                }  
            }  
            // 每一行以换行符结束  
            file << std::endl;  
        }  
        file.close();  
        std::cout << "Appended matrix to " << fileName << std::endl;  
    }  
    else  
    {  
        std::cout << "Failed to open file: " << fileName << std::endl;  
    }  
}  



// 将矩阵输出到文件中，支持文件追加写入  
void writeMatrixToCsvV2(double *dA, long ldA, long rows, long cols, const std::string &fileName, const std::string &matrixName)  
{  
    double matrix;  

    // 以 "追加写入" 模式打开文件  
    std::ofstream file(fileName, std::ios::app);  

    if (file.is_open())  
    {  
        // 如果需要，每次追加写入前可标明当前矩阵的名称  
        file << matrixName << std::endl;  

        // 设置浮点数输出固定为 4 位小数 + 列对齐  
        file << std::fixed << std::setprecision(3);  

        // 假设每列宽度固定为 10，可以调整此值以适配更宽或更小的输出  
        const int columnWidth = 6;  

        for (long i = 0; i < rows; i++)  
        {  
            for (long j = 0; j < cols; j++)  
            {  
                // 从设备内存中复制一个元素到主机  
                hipMemcpy(&matrix, dA + i + j * ldA, sizeof(double), hipMemcpyDeviceToHost);  

                // 输出当前元素，使用宽度和右对齐  
                file << std::right << std::setw(columnWidth) << matrix;  

                // 如果不是当前行的最后一个元素，添加逗号对齐  
                if ((cols - 1) != j)  
                {  
                    file << " ";  
                }  
            }  
            // 每一行以换行符结束  
            file << std::endl;  
        }  
        file.close();  
        std::cout << "Appended matrix to " << fileName << std::endl;  
    }  
    else  
    {  
        std::cout << "Failed to open file: " << fileName << std::endl;  
    }  
}  




void loadMatrixFromCsvToGpu(const std::string &fileName,  
                            float *&dA,  
                            int rows,  
                            int cols)  
{  
    // 临时存储矩阵数据（CPU端）  
    std::vector<std::vector<float>> matrixData;  
    std::ifstream file(fileName);  

    if (!file.is_open()) {  
        std::cerr << "Failed to open file: " << fileName << std::endl;  
        dA = nullptr;  
        return;  
    }  

    std::string line;  

    // 读取矩阵数据（按行读取并解析为浮点数）  
    while (std::getline(file, line)) {  
        if (line.empty()) {  
            continue; // 忽略空行  
        }  

        // 使用 istringstream 流来读取浮点数，能自动处理多个空格  
        std::istringstream lineStream(line);  
        std::vector<float> row;  
        float value;  

        // 利用 operator>> 逐个提取浮点数  
        while (lineStream >> value) {  
            row.push_back(value);  
        }  

        // 如果这一行解析后没有数字，可以选择忽略  
        if (!row.empty()) {  
            matrixData.push_back(row);  
        }  
    }  

    file.close();  

    // 获取矩阵形状  
    rows = matrixData.size();  
    cols = (rows > 0) ? matrixData[0].size() : 0;  

    if (rows == 0 || cols == 0) {  
        std::cerr << "Invalid matrix dimensions in file: " << fileName << std::endl;  
        dA = nullptr;  
        return;  
    }  

    // 分配主机内存，并将数据填充到线性内存中  
    std::vector<float> hostMatrix(rows * cols);  
    for (int i = 0; i < rows; ++i) {  
        for (int j = 0; j < cols; ++j) {  
            // hostMatrix[i * cols + j] = matrixData[i][j];  
            hostMatrix[j * rows + i] = matrixData[i][j];  
        }  
    }  

    // 分配 GPU 内存  
    hipError_t err = hipMalloc(&dA, rows * cols * sizeof(float));  
    if (err != hipSuccess) {  
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;  
        dA = nullptr;  
        return;  
    }  

    // 拷贝数据到 GPU  
    err = hipMemcpy(dA, hostMatrix.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);  
    if (err != hipSuccess) {  
        std::cerr << "CUDA memcpy to device failed: " << hipGetErrorString(err) << std::endl;  
        hipFree(dA);  
        dA = nullptr;  
        return;  
    }  

    std::cout << "Successfully loaded matrix with dimensions (" << rows << "x" << cols << ") to GPU." << std::endl;  
}  